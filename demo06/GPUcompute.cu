#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <windows.h>

#define w 5000

int gettimeofday(struct timeval* tp, void* tzp)
{
    time_t clock;
    struct tm tm;
    SYSTEMTIME wtm;
    GetLocalTime(&wtm);
    tm.tm_year = wtm.wYear - 1900;
    tm.tm_mon = wtm.wMonth - 1;
    tm.tm_mday = wtm.wDay;
    tm.tm_hour = wtm.wHour;
    tm.tm_min = wtm.wMinute;
    tm.tm_sec = wtm.wSecond;
    tm.tm_isdst = -1;
    clock = mktime(&tm);
    tp->tv_sec = clock;
    tp->tv_usec = wtm.wMilliseconds * 1000;
    return (0);
}

struct Matrix
{
    int width;
    int height;
    float* elements;
};

__device__ float getElement(Matrix* A, int row, int col)
{
    return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix* A, int row, int col, float value)
{
    A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix* A, Matrix* B, Matrix* C)
{
    float Cvalue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < A->width; ++i)
    {
        Cvalue += getElement(A, row, i) * getElement(B, i, col);
    }
    setElement(C, row, col, Cvalue);
}

int main()
{
    int width = w;
    int height = w;

    Matrix* A, * B, * C;

    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));

    int nBytes = width * height * sizeof(float);

    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;

    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y);

    struct timeval t1, t2;
    gettimeofday(&t1, NULL);
    double timeuse;

    matMulKernel << < gridSize, blockSize >> > (A, B, C);

    hipDeviceSynchronize();

    gettimeofday(&t2, NULL);
    timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Use Time:%fs\n", timeuse);
    system("pause");
    return 0;
}
