#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>


__global__ void MatMul(int* M, int* N, int* P, int width)
{
	int x = threadIdx.x;
	int y = threadIdx.y;

	float Pervalue = 0;

	float elem1 = 0.0, elem2 = 0.0, value = 0.0;
	for (int i = 0; i < width; i++)
	{
		elem1 = M[y * width + i];//ȡM�����һ��
		elem2 = N[i * width + x];//ȡN�����һ��

		value += elem1 * elem2;//���
	}

	P[y * width + x] = value;
}

int main()
{
	const int ND = 30; //�㶨����ı��� ND=30
	int a[ND][ND], b[ND][ND], c[ND][ND];
	int* M, * N, * P;	//int��ָ��

	int width = ND;	//������
	int NUM = 900;	//
	dim3 blockSize(ND, ND); 

	hipEvent_t start, stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//�豸���ڴ����
	hipMalloc((void**)&M, ND * ND * sizeof(int));
	hipMalloc((void**)&N, ND * ND * sizeof(int));
	hipMalloc((void**)&P, ND * ND * sizeof(int));

	//��ʼ��
	for (int i = 0; i < ND; i++)
	{
		for (int j = 0; j < ND; j++)
		{
			a[i][j] = 2;
			b[i][j] = 3;
		}
	}

	int Size = ND * ND;

	//���ݿ������������豸
	hipMemcpy(M, a, Size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(N, b, Size * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	MatMul <<<1, blockSize>>> (M, N, P, width);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(c, P, Size * sizeof(int), hipMemcpyDeviceToHost);

	//printf("c0 = %d \n", c[0][0]);

	for (int i = 0; i < ND; i++)
	{
		for (int j = 0; j < ND; j++)
			printf("%d ", c[i][j]);
		printf("\n");
	}

	//�ͷ��豸�ڴ�
	hipFree(M);
	hipFree(N);
	hipFree(P);

	return 0;
}