#include "hip/hip_runtime.h"
#include "stdio.h"

/*
���η�__global__������������ǽ����CPU�е��ã���GPU�н���ִ�С�
����˺����������ں˺�����
*/
__global__ void hello_world_from_gpu(void)
{
	printf("Hello World from GPU\n");
	return;
}

int main(void)
{
	printf("Hello World from CPU\n");

	hello_world_from_gpu << <1, 5 >> > ();
	/*
	���ؼ�������Ĳ�������������ص�ִ�����ã���������ʹ�ö����߳���ִ���ں˺�����
	�ڱ���������5��GPU�̱߳�ϵͳ�����á�
	*/
	hipDeviceReset();
	/*
	ִ����ɺ����hipDeviceReset()�����ͷź�����뵱ǰ����������ص���Դ��
	*/
	return 0;
}